#include "hip/hip_runtime.h"
#include "option/european.h"

#include <hiprand/hiprand_kernel.h>

#include <iostream>
#include <stdio.h>

#define THREADBLOCK_SIZE 1024

#define r 0.03

__global__ void InitRandomStates(hiprandState_t* devStates) {
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  /* Each thread gets same seed, a different sequence
            number, no offset */
  hiprand_init(1234, id, 0, &devStates[id]);
}

__device__ void GenerateSamplePath(VanillaEuropean option, double* path,
    const int timeSteps, hiprandState_t* state) {
  double St = option.GetS0();
  double sigma = option.GetSigma();
  double T = option.GetTTM();
  int m = timeSteps;
  for (int k = 0; k < m; k++) {
    path[k] = St * exp((r - sigma*sigma * 0.5) * (T/m) + sigma * sqrt(T/m) 
        * hiprand_normal_double(state));
    St = path[k];
  }
}

__global__ void PriceByMC(VanillaEuropean* options, double* optionValues, 
    const int optionsNum, const long simNum, const int timeSteps,
    hiprandState_t* devStates) {
  int bid = blockIdx.x;
  int tid = threadIdx.x;
  int id = tid + bid * blockDim.x;

  int simNo = tid;
  int i = 0;

  __shared__ double payoffs[THREADBLOCK_SIZE];
  double threadPayoff = 0.0;

  hiprandState_t* state = &devStates[id];

  if (bid >= optionsNum) return;

  VanillaEuropean option = options[bid];
  double* path = new double[timeSteps];

  while (simNo < simNum) {
    // Generate sample path of this option
    GenerateSamplePath(option, path, timeSteps, state);
    threadPayoff = (i * threadPayoff + option.Payoff(path, timeSteps)) 
      / (i + 1.0);
    simNo += THREADBLOCK_SIZE;
    i++;
  }
  payoffs[tid] = threadPayoff;
  __syncthreads();

  if (tid == 0) {
    double avg = 0.0;
    for (int j = 0; j < THREADBLOCK_SIZE; ++j) {
      avg = (j * avg + payoffs[j]) / (j + 1.0);
    }
    optionValues[bid] = exp(-r * option.GetTTM()) * avg;
  }
}

int main() {
  const bool isCall = true;
  const double strike = 100.0;
  const double s0 = 100.0;
  const double sigma = 0.2;
  const double ttm = 1.0 / 12.0; // 1 month

  const int optionsNum = 1;
  const long simNum = 100000;
  const int timeSteps = 300;

  VanillaEuropean options[optionsNum];
  options[0] = VanillaEuropean(isCall, strike, s0, sigma, ttm);

  VanillaEuropean* dev_options;
  hipMalloc((void**) &dev_options, sizeof(VanillaEuropean) * optionsNum);

  hipMemcpy(dev_options, options, optionsNum * sizeof(VanillaEuropean),
      hipMemcpyHostToDevice);

  double optionValues[optionsNum];

  double* dev_optionValues;
  hipMalloc((void**) &dev_optionValues, sizeof(double) * optionsNum);

  const int totalThreads = 1024 * optionsNum;
  hiprandState_t* devStates;
  hipMalloc((void**) &devStates, totalThreads * sizeof(hiprandState_t));

  InitRandomStates<<<optionsNum, 1024>>>(devStates);

  PriceByMC<<<optionsNum, 1024>>>(dev_options, dev_optionValues, optionsNum,
      simNum, timeSteps, devStates);

  hipMemcpy(optionValues, dev_optionValues, sizeof(double) * optionsNum,
      hipMemcpyDeviceToHost);

  std::cout << "Option value = " << optionValues[0] << std::endl;
  std::cout << "By BS Forumla = " << options[0].PriceByBSFormula(r) 
    << std::endl;

  hipFree(dev_options);
  hipFree(dev_optionValues);
  hipFree(devStates);
}
