#include "hip/hip_runtime.h"
#include "option/european.h"

#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime_api.h>

#include <iostream>
#include <stdio.h>

#define THREADBLOCK_SIZE 512
/* #define THREADBLOCK_SIZE 2 */

#define r 0.03

__global__ void InitRandomStates(hiprandState_t* devStates) {
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  /* Each thread gets same seed, a different sequence
            number, no offset */
  hiprand_init(1234, id, 0, &devStates[id]);
}

__global__ void InitRandomStatesQuasi(hiprandStateScrambledSobol32_t* devStates,
    hiprandDirectionVectors32_t* directionVectors, unsigned int* scrambleConstants) {
  int id = threadIdx.x + blockIdx.x * blockDim.x;

  // Use the same timeSteps dimensions, but offset each thread by ?
  for (int i = 0; i < 300; i++) {
    unsigned int offset = id * ((10000 / blockDim.x) + 1);
    /* printf("blockDim.x = %d    blockIdx.x = %d\n", blockDim.x, blockIdx.x); */
    /* printf("rand init    id = %d    dim = %d    offset = %u    state idx = %d\n", id, i, offset, 300 * id + i); */
    hiprand_init(directionVectors[i], scrambleConstants[i], offset, &devStates[300 * id + i]);
  }
}

__device__ void GenerateSamplePath(VanillaEuropean option, double* path,
    const int timeSteps, hiprandState_t* state) {
  double St = option.GetS0();
  double sigma = option.GetSigma();
  double T = option.GetTTM();
  int m = timeSteps;
  for (int k = 0; k < m; k++) {
    const double z = hiprand_normal_double(state);
    path[k] = St * exp((r - sigma*sigma * 0.5) * (T/m) + sigma * sqrt(T/m) 
        * z);
    St = path[k];
    /* if (threadIdx.x == 0 && blockIdx.x == 0) */
    /*   printf("z = %f  St = %f\n", z, St); */
  }
}

__device__ void GenerateSamplePathQuasi(VanillaEuropean option, double* path,
    const int timeSteps, hiprandStateScrambledSobol32_t* states) {
  double St = option.GetS0();
  double sigma = option.GetSigma();
  double T = option.GetTTM();
  int m = timeSteps;
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  for (int k = 0; k < m; k++) {
    const double z = hiprand_normal_double(&states[k]);
    if (&path[k] == 0x0)
      printf("ERROR: Accessing random state index = %d\n&path = %p\n===\n", 300 * id + k, path);
    path[k] = St * exp((r - sigma*sigma * 0.5) * (T/m) + sigma * sqrt(T/m) 
        * z);
    St = path[k];
    /* if (threadIdx.x == 0 && blockIdx.x == 0) */
    /*   printf("generator addr = %p  z = %f  St = %f\n", &states[k], z, St); */
  }
}

__global__ void PriceByMC(VanillaEuropean* options, double* optionValues, 
    double* optionDeltas, const int optionsNum, const long simNum, 
    const int timeSteps, hiprandState_t* devStates, double* dev_paths) {
  int bid = blockIdx.x;
  int tid = threadIdx.x;
  int id = tid + bid * blockDim.x;

  int simNo = tid;
  int i = 0;

  __shared__ double payoffs[THREADBLOCK_SIZE];
  __shared__ double deltas[THREADBLOCK_SIZE];
  double threadPayoff = 0.0;
  double threadDelta = 0.0;

  hiprandState_t* state = &devStates[id];

  if (bid >= optionsNum) return;

  VanillaEuropean option = options[bid];
  double* path = &dev_paths[300 * id];

  while (simNo < simNum) {
    // Generate sample path of this option
    GenerateSamplePath(option, path, timeSteps, state);
    threadPayoff = (i * threadPayoff + option.Payoff(path, timeSteps)) 
      / (i + 1.0);
    /* if (threadIdx.x == 96) */
    /*   printf("last path = %f  avg payoff so far = %f\n", path[timeSteps-1], threadPayoff); */
    double dY_dS0 = (path[timeSteps - 1] / option.GetS0())
      * (path[timeSteps - 1] > option.GetStrike() ? 1.0 : 0.0);
    threadDelta = (i * threadDelta + dY_dS0) / (i + 1.0);
    simNo += THREADBLOCK_SIZE;
    i++;
  }
  payoffs[tid] = threadPayoff;
  deltas[tid] = threadDelta;
  __syncthreads();
  /* printf("FINAL THREAD PAYOFF: %f\n", threadPayoff); */

  if (tid == 0) {
    double avg = 0.0;
    double deltaAvg = 0.0; 
    for (int j = 0; j < THREADBLOCK_SIZE; ++j) {
      avg = (j * avg + payoffs[j]) / (j + 1.0);
      deltaAvg = (j * deltaAvg + deltas[j]) / (j + 1.0);
    }
    optionValues[bid] = exp(-r * option.GetTTM()) * avg;
    optionDeltas[bid] = exp(-r * option.GetTTM()) * deltaAvg;
  }
}

__global__ void PriceByMCQuasi(VanillaEuropean* options, double* optionValues, 
    double* optionDeltas, const int optionsNum, const long simNum, 
    const int timeSteps, hiprandStateScrambledSobol32_t* devStates, double* dev_paths) {
  int bid = blockIdx.x;
  int tid = threadIdx.x;
  int id = tid + bid * blockDim.x;

  int simNo = tid;
  int i = 0;

  __shared__ double payoffs[THREADBLOCK_SIZE];
  __shared__ double deltas[THREADBLOCK_SIZE];
  double threadPayoff = 0.0;
  double threadDelta = 0.0;

  hiprandStateScrambledSobol32_t* states = &devStates[300 * id];

  if (bid >= optionsNum) return;

  VanillaEuropean option = options[bid];
  double* path = &dev_paths[300 * id];

  while (simNo < simNum) {
    // Generate sample path of this option
    GenerateSamplePathQuasi(option, path, timeSteps, states);
    threadPayoff = (i * threadPayoff + option.Payoff(path, timeSteps)) 
      / (i + 1.0);
    /* if (id < 5) */
    /*   printf("id = %d  last path = %f  avg payoff so far = %f\n", id, path[timeSteps-1], threadPayoff); */
    double dY_dS0 = (path[timeSteps - 1] / option.GetS0())
      * (path[timeSteps - 1] > option.GetStrike() ? 1.0 : 0.0);
    threadDelta = (i * threadDelta + dY_dS0) / (i + 1.0);
    simNo += THREADBLOCK_SIZE;
    i++;
  }
  payoffs[tid] = threadPayoff;
  deltas[tid] = threadDelta;
  __syncthreads();
  /* printf("FINAL THREAD PAYOFF: %f\n", threadPayoff); */

  if (tid == 0) {
    double avg = 0.0;
    double deltaAvg = 0.0; 
    for (int j = 0; j < THREADBLOCK_SIZE; ++j) {
      avg = (j * avg + payoffs[j]) / (j + 1.0);
      deltaAvg = (j * deltaAvg + deltas[j]) / (j + 1.0);
    }
    optionValues[bid] = exp(-r * option.GetTTM()) * avg;
    optionDeltas[bid] = exp(-r * option.GetTTM()) * deltaAvg;
  }
}

int main() {
  const bool isCall = true;
  const double strike = 100.0;
  const double s0 = 100.0;
  const double sigma = 0.2;
  const double ttm = 1.0 / 12.0; // 1 month

  const int optionsNum = 10;
  const long simNum = 30000;
  const int timeSteps = 300;

  VanillaEuropean options[optionsNum];
  for (int i = 0; i < optionsNum; ++i) {
    options[i] = VanillaEuropean(isCall, strike, s0, sigma, ttm);
  }

  VanillaEuropean* dev_options;
  hipMalloc((void**) &dev_options, sizeof(VanillaEuropean) * optionsNum);

  hipMemcpy(dev_options, options, optionsNum * sizeof(VanillaEuropean),
      hipMemcpyHostToDevice);

  double optionValues[optionsNum];
  double optionDeltas[optionsNum];

  double* dev_optionValues;
  double* dev_optionDeltas;
  hipMalloc((void**) &dev_optionValues, sizeof(double) * optionsNum);
  hipMalloc((void**) &dev_optionDeltas, sizeof(double) * optionsNum);

  const int totalThreads = THREADBLOCK_SIZE * optionsNum;

  double* dev_paths;
  checkCudaErrors(hipMalloc((void**) &dev_paths, totalThreads * timeSteps * sizeof(double)));

  hiprandState_t* devStates;
  hipMalloc((void**) &devStates, totalThreads * sizeof(hiprandState_t));

  // Each thread has timeSteps states
  hiprandStateScrambledSobol32_t* devQuasiStates;
  checkCudaErrors(hipMalloc((void**) &devQuasiStates,
      totalThreads * timeSteps * sizeof(hiprandStateScrambledSobol32_t)));

  hiprandDirectionVectors32_t* directionVectors;
  hiprandGetDirectionVectors32(&directionVectors,
      HIPRAND_SCRAMBLED_DIRECTION_VECTORS_32_JOEKUO6);

  // We need timeSteps dimensions (dimensions are shared but with each thread having an offset)
  hiprandDirectionVectors32_t* dev_directionVectors;
  checkCudaErrors(hipMalloc((void**) &dev_directionVectors,
      timeSteps * sizeof(hiprandDirectionVectors32_t)));
  /* printf("%d\n", totalThreads * timeSteps); */
  /* printf("%lu\n", totalThreads * timeSteps * sizeof(hiprandDirectionVectors32_t)); */

  checkCudaErrors(hipMemcpy(dev_directionVectors, directionVectors,
      timeSteps * sizeof(hiprandDirectionVectors32_t), hipMemcpyHostToDevice));

  unsigned int* scrambleConstants;
  hiprandGetScrambleConstants32(&scrambleConstants);
  unsigned int* dev_scrambleConstants;
  checkCudaErrors(hipMalloc((void**) &dev_scrambleConstants,
        timeSteps * sizeof(unsigned int)));
  checkCudaErrors(hipMemcpy(dev_scrambleConstants, scrambleConstants,
        timeSteps * sizeof(unsigned int), hipMemcpyHostToDevice));

  InitRandomStates<<<optionsNum, THREADBLOCK_SIZE>>>(devStates);
  InitRandomStatesQuasi<<<optionsNum, THREADBLOCK_SIZE>>>(devQuasiStates,
      dev_directionVectors, dev_scrambleConstants);
  getLastCudaError("Initialisation failed\n");

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  PriceByMCQuasi<<<optionsNum, THREADBLOCK_SIZE>>>(dev_options, dev_optionValues, 
      dev_optionDeltas, optionsNum,
      simNum, timeSteps, devQuasiStates, dev_paths);
  hipEventRecord(stop);
  hipDeviceSynchronize();
  getLastCudaError("Quasi kernel failed\n");

  /* hipEventRecord(start); */
  /* PriceByMC<<<optionsNum, THREADBLOCK_SIZE>>>(dev_options, dev_optionValues, */ 
  /*     dev_optionDeltas, optionsNum, */
  /*     simNum, timeSteps, devStates, dev_paths); */
  /* hipEventRecord(stop); */
  /* hipDeviceSynchronize(); */
  /* getLastCudaError("Normal kernel failed\n"); */

  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Took: %fms\n", milliseconds);

  hipMemcpy(optionValues, dev_optionValues, sizeof(double) * optionsNum,
      hipMemcpyDeviceToHost);
  hipMemcpy(optionDeltas, dev_optionDeltas, sizeof(double) * optionsNum,
      hipMemcpyDeviceToHost);

  std::cout << "S(0) = " << options[0].GetS0() << std::endl;
  std::cout << "K = " << options[0].GetStrike() << std::endl;
  std::cout << "TTM = " << options[0].GetTTM() << std::endl;
  std::cout << "sigma = " << options[0].GetSigma() << std::endl;
  std::cout << "isCall? " << options[0].IsCall() << std::endl;
  std::cout << "r = " << r << std::endl;

  std::cout << std::endl << "=== Calculated ===" << std::endl;
  for (int i = 0; i < optionsNum; ++i) {
    std::cout << "Option " << i << "\n---------" << std::endl;
    std::cout << "Option value = " << optionValues[i] << std::endl;
    std::cout << "Delta = " << optionDeltas[i] << std::endl;
    std::cout << "BS Forumla value = " << options[i].PriceByBSFormula(r) 
      << std::endl;
    std::cout << "Absolute error = " << abs(options[i].PriceByBSFormula(r) - optionValues[i]) << std::endl << std::endl;
  }

  hipFree(dev_options);
  hipFree(dev_optionValues);
  hipFree(devStates);
  hipFree(devQuasiStates);
  hipFree(dev_directionVectors);
  hipFree(dev_paths);
  hipEventDestroy(start);
  hipEventDestroy(stop);
}
