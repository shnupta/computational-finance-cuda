#include "common.h"
#include "option.h"
#include "black_scholes.h"
#include "binomial_gpu.h"
#include "binomial_cpu.h"
#include <iostream>
#include <cmath>
#include <chrono>

#include <hip/hip_runtime_api.h>

// Generate a uniformly distributed random float in the range [low, high]
double UniformRandom(double low, double high) {
  double r = static_cast<double>(rand()) / static_cast<double>(RAND_MAX);
  return (1 - r) * low + r * high;
}

int main() {
  const int numOptions = OPTIONS_NUM;
  const int timeSteps = TIME_STEPS;

  float elapsedTimeGPU;
  hipEvent_t startGpu, endGpu;
  checkCudaErrors(hipEventCreate(&startGpu));
  checkCudaErrors(hipEventCreate(&endGpu));

  EuropeanOption options[numOptions];
  double callValueBS[numOptions];
  double callValueCPU[numOptions];
  double callValueGPU[numOptions];

  std::cout << "Generating input option data...\n";

  srand(1896);

  // Generate input option data
  for (int i = 0; i < numOptions; i++) {
    options[i].S = UniformRandom(5.0f, 30.0f); 
    options[i].K = UniformRandom(1.0f, 100.0f);
    options[i].T = UniformRandom(0.25f, 10.0f);
    options[i].R = 0.06f;
    options[i].V = 0.10f;
    // Calculate the value of this option using Black-Scholes formula for comparison later
    BlackScholesCall(callValueBS[i], options[i]);
  }

  std::cout << "Generated " << numOptions << " options.\n";
  std::cout << "Running over " << timeSteps << " time steps.\n\n";

  std::cout << "Running GPU kernel...\n";
  checkCudaErrors(hipEventRecord(startGpu, 0));
  BinomialPricingGPU(callValueGPU, options, numOptions);
  checkCudaErrors(hipEventRecord(endGpu, 0));
  checkCudaErrors(hipEventSynchronize(endGpu));

  checkCudaErrors(hipEventElapsedTime(&elapsedTimeGPU, startGpu, endGpu));
  std::cout << "Time taken: " << elapsedTimeGPU << " ms\n\n";

  std::cout << "Running CPU version...\n";
  std::chrono::steady_clock::time_point startCpu = std::chrono::steady_clock::now();
  for (int i = 0; i < numOptions; i++) {
    BinomialPricingCPU(callValueCPU[i], options[i]);
  }
  std::chrono::steady_clock::time_point endCpu = std::chrono::steady_clock::now();
  std::cout << "Time taken: " << std::chrono::duration_cast<std::chrono::milliseconds>(endCpu - startCpu).count() << " ms\n";

  return 0;
}
